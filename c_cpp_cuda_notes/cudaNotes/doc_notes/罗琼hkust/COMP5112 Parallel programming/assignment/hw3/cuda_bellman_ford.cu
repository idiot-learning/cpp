/**
 * Name: Chao, WANG 
 * Student id:20365793
 * ITSC email:cwangbh@connect.ust.hk
 */
/*
 * This is a CUDA version of bellman_ford algorithm
 * Compile: nvcc -std=c++11 -arch=sm_52 -o cuda_bellman_ford cuda_bellman_ford.cu
 * Run: ./cuda_bellman_ford <input file> <number of blocks per grid> <number of threads per block>, you will find the output file 'output.txt'
 * */

#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <iomanip>
#include <cstring>
#include <sys/time.h>

#include <hip/hip_runtime.h>


using std::string;
using std::cout;
using std::endl;

#define INF 1000000

/*
 * This is a CHECK function to check CUDA calls
 */
#define CHECK(call)                                                            \
		{                                                                              \
	const hipError_t error = call;                                            \
	if (error != hipSuccess)                                                  \
	{                                                                          \
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
		fprintf(stderr, "code: %d, reason: %s\n", error,                       \
				hipGetErrorString(error));                                    \
				exit(1);                                                               \
	}                                                                          \
		}


/**
 * utils is a namespace for utility functions
 * including I/O (read input file and print results) and matrix dimension convert(2D->1D) function
 */
namespace utils {
int N; //number of vertices
int *mat; // the adjacency matrix

void abort_with_error_message(string msg) {
	std::cerr << msg << endl;
	abort();
}

//translate 2-dimension coordinate to 1-dimension
int convert_dimension_2D_1D(int x, int y, int n) {
	return x * n + y;
}

int read_file(string filename) {
	std::ifstream inputf(filename, std::ifstream::in);
	if (!inputf.good()) {
		abort_with_error_message("ERROR OCCURRED WHILE READING INPUT FILE");
	}
	inputf >> N;
	//input matrix should be smaller than 20MB * 20MB (400MB, we don't have too much memory for multi-processors)
	assert(N < (1024 * 1024 * 20));
	mat = (int *) malloc(N * N * sizeof(int));
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++) {
			inputf >> mat[convert_dimension_2D_1D(i, j, N)];
		}
	return 0;
}

int print_result(bool has_negative_cycle, int *dist) {
	std::ofstream outputf("output.txt", std::ofstream::out);
	if (!has_negative_cycle) {
		for (int i = 0; i < N; i++) {
			if (dist[i] > INF)
				dist[i] = INF;
			outputf << dist[i] << '\n';
		}
		outputf.flush();
	} else {
		outputf << "FOUND NEGATIVE CYCLE!" << endl;
	}
	outputf.close();
	return 0;
}
}//namespace utils

// you may add some helper/kernel functions here.

// kernel functions here
// relax the edge
__global__ void updateDist(int *d_dist, int *d_mat, int n, bool *d_hasnc)
{
	// allocate local variables
	int idx = blockDim.x*blockIdx.x + threadIdx.x;   // thread id
	int u,v;                                         // used for loops
	int weight;
	int lenskip = blockDim.x*gridDim.x;              // jump length for coalesce mem visiting.
	// bellman ford edge relaxation

	*d_hasnc = false;
	for (u = 0; u < n; u++)
	{
		for (v = idx; v < n; v=v+lenskip)
		{
			weight = d_mat[u*n+v];
			if (weight<INF)
				if (d_dist[u]+weight < d_dist[v])
				{
					d_dist[v] = d_dist[u] + weight;
					*d_hasnc = true;
				}
		}
	}
}

// cudaDeviceSynchronize
// kernel function: find negative cycle.
__global__ void negativeDetect(int *d_dist, int *d_mat, const int n, bool *d_hasnc)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;   // thread id
	int u,v;
	int weight;
	int lenskip = blockDim.x * gridDim.x;
	
	*d_hasnc = false;
	for (u = 0; u < n; u++)
	{
		for (v = idx; v < n; v=v+lenskip)
		{
			weight = d_mat[u*n+v];
			if (weight<INF)
				if (d_dist[u]+weight < d_dist[v])
					*d_hasnc = true;
		}
	}
}


//

/**
 * Bellman-Ford algorithm. Find the shortest path from vertex 0 to other vertices.
 * @param blockPerGrid number of blocks per grid
 * @param threadsPerBlock number of threads per block
 * @param n input size
 * @param *mat input adjacency matrix
 * @param *dist distance array
 * @param *has_negative_cycle a bool variable to recode if there are negative cycles
 */
void bellman_ford(int blocksPerGrid, int threadsPerBlock, int n, int *mat, int *dist, bool *has_negative_cycle) {
	//------your code starts from here------
	// init dist on CPU
	int i;
	bool haschange;
	
	for (i = 1; i < n; i++)
	{
		dist[i] = INF;
	}
	dist[0] = 0;
	
	// create global mem in GPU
	int *d_dist, *d_mat;
	bool *d_hasnc;
	
	*has_negative_cycle = true;
	hipMalloc(&d_dist, sizeof(int)*n);
	hipMalloc(&d_mat, sizeof(int)*n*n);
	hipMalloc(&d_hasnc, sizeof(bool));
	// memcopy from CPU to GPU
	hipMemcpy(d_dist, dist, sizeof(int)*n, hipMemcpyHostToDevice);
	hipMemcpy(d_mat, mat, sizeof(int)*n*n, hipMemcpyHostToDevice);
	hipMemcpy(d_hasnc, has_negative_cycle, sizeof(bool), hipMemcpyHostToDevice);
	// Call device function to proceed.
	for (i = 0; i < n; i++)
	{
		haschange = false;
		updateDist<<<blocksPerGrid, threadsPerBlock>>>(d_dist, d_mat, n, d_hasnc);
		hipMemcpy(&haschange,d_hasnc, sizeof(bool),hipMemcpyDeviceToHost);
		if (!haschange)
		{
			*has_negative_cycle = false;
			break;
		}
	}
	// 
	if (*has_negative_cycle)
	{
		negativeDetect<<<blocksPerGrid, threadsPerBlock>>>(d_dist, d_mat, n, d_hasnc);
		hipMemcpy(has_negative_cycle, d_hasnc, sizeof(bool), hipMemcpyDeviceToHost);
	}
	*has_negative_cycle = false;
	// memcopy from GPU to CPU
    hipMemcpy(dist, d_dist, sizeof(int)*n, hipMemcpyDeviceToHost);	
	
	// free memory on GPU
	hipFree(d_dist);
	hipFree(d_mat);
	hipFree(d_hasnc);
	// end of the code
}

int main(int argc, char **argv) {
	if (argc <= 1) {
		utils::abort_with_error_message("INPUT FILE WAS NOT FOUND!");
	}
	if (argc <= 3) {
		utils::abort_with_error_message("blocksPerGrid or threadsPerBlock WAS NOT FOUND!");
	}

	string filename = argv[1];
	int blocksPerGrid = atoi(argv[2]);
	int threadsPerBlock = atoi(argv[3]);

	int *dist;
	bool has_negative_cycle = false;


	assert(utils::read_file(filename) == 0);
	dist = (int *) calloc(sizeof(int), utils::N);


	//time counter
	timeval start_wall_time_t, end_wall_time_t;
	float ms_wall;
	hipDeviceReset();
	//start timer
	gettimeofday(&start_wall_time_t, nullptr);
	//bellman-ford algorithm
	bellman_ford(blocksPerGrid, threadsPerBlock, utils::N, utils::mat, dist, &has_negative_cycle);
	CHECK(hipDeviceSynchronize());
	//end timer
	gettimeofday(&end_wall_time_t, nullptr);
	ms_wall = ((end_wall_time_t.tv_sec - start_wall_time_t.tv_sec) * 1000 * 1000
			+ end_wall_time_t.tv_usec - start_wall_time_t.tv_usec) / 1000.0;

	std::cerr.setf(std::ios::fixed);
	std::cerr << std::setprecision(6) << "Time(s): " << (ms_wall/1000.0) << endl;
	utils::print_result(has_negative_cycle, dist);
	free(dist);
	free(utils::mat);

	return 0;
}
