/**
 * Name:
 * Student id:
 * ITSC email:
 */
/*
 * This is a CUDA version of bellman_ford algorithm
 * Compile: nvcc -std=c++11 -arch=sm_52 -o cuda_bellman_ford cuda_bellman_ford.cu
 * Run: ./cuda_bellman_ford <input file> <number of blocks per grid> <number of threads per block>, you will find the output file 'output.txt'
 * */

#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <iomanip>
#include <cstring>
#include <sys/time.h>

#include <hip/hip_runtime.h>


using std::string;
using std::cout;
using std::endl;

#define INF 1000000

/*
 * This is a CHECK function to check CUDA calls
 */
#define CHECK(call)                                                            \
		{                                                                              \
	const hipError_t error = call;                                            \
	if (error != hipSuccess)                                                  \
	{                                                                          \
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
		fprintf(stderr, "code: %d, reason: %s\n", error,                       \
				hipGetErrorString(error));                                    \
				exit(1);                                                               \
	}                                                                          \
		}


/**
 * utils is a namespace for utility functions
 * including I/O (read input file and print results) and matrix dimension convert(2D->1D) function
 */
namespace utils {
int N; //number of vertices
int *mat; // the adjacency matrix

void abort_with_error_message(string msg) {
	std::cerr << msg << endl;
	abort();
}

//translate 2-dimension coordinate to 1-dimension
int convert_dimension_2D_1D(int x, int y, int n) {
	return x * n + y;
}

int read_file(string filename) {
	std::ifstream inputf(filename, std::ifstream::in);
	if (!inputf.good()) {
		abort_with_error_message("ERROR OCCURRED WHILE READING INPUT FILE");
	}
	inputf >> N;
	//input matrix should be smaller than 20MB * 20MB (400MB, we don't have too much memory for multi-processors)
	assert(N < (1024 * 1024 * 20));
	mat = (int *) malloc(N * N * sizeof(int));
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++) {
			inputf >> mat[convert_dimension_2D_1D(i, j, N)];
		}
	return 0;
}

int print_result(bool has_negative_cycle, int *dist) {
	std::ofstream outputf("output.txt", std::ofstream::out);
	if (!has_negative_cycle) {
		for (int i = 0; i < N; i++) {
			if (dist[i] > INF)
				dist[i] = INF;
			outputf << dist[i] << '\n';
		}
		outputf.flush();
	} else {
		outputf << "FOUND NEGATIVE CYCLE!" << endl;
	}
	outputf.close();
	return 0;
}
}//namespace utils

// you may add some helper/kernel functions here.

/**
 * Bellman-Ford algorithm. Find the shortest path from vertex 0 to other vertices.
 * @param blockPerGrid number of blocks per grid
 * @param threadsPerBlock number of threads per block
 * @param n input size
 * @param *mat input adjacency matrix
 * @param *dist distance array
 * @param *has_negative_cycle a bool variable to recode if there are negative cycles
 */
void bellman_ford(int blocksPerGrid, int threadsPerBlock, int n, int *mat, int *dist, bool *has_negative_cycle) {
	//------your code starts from here------

	//------end of your code------
}

int main(int argc, char **argv) {
	if (argc <= 1) {
		utils::abort_with_error_message("INPUT FILE WAS NOT FOUND!");
	}
	if (argc <= 3) {
		utils::abort_with_error_message("blocksPerGrid or threadsPerBlock WAS NOT FOUND!");
	}

	string filename = argv[1];
	int blockPerGrid = atoi(argv[2]);
	int threadsPerBlock = atoi(argv[3]);

	int *dist;
	bool has_negative_cycle = false;


	assert(utils::read_file(filename) == 0);
	dist = (int *) calloc(sizeof(int), utils::N);


	//time counter
	timeval start_wall_time_t, end_wall_time_t;
	float ms_wall;
	hipDeviceReset();
	//start timer
	gettimeofday(&start_wall_time_t, nullptr);
	//bellman-ford algorithm
	bellman_ford(blockPerGrid, threadsPerBlock, utils::N, utils::mat, dist, &has_negative_cycle);
	CHECK(hipDeviceSynchronize());
	//end timer
	gettimeofday(&end_wall_time_t, nullptr);
	ms_wall = ((end_wall_time_t.tv_sec - start_wall_time_t.tv_sec) * 1000 * 1000
			+ end_wall_time_t.tv_usec - start_wall_time_t.tv_usec) / 1000.0;

	std::cerr.setf(std::ios::fixed);
	std::cerr << std::setprecision(6) << "Time(s): " << (ms_wall/1000.0) << endl;
	utils::print_result(has_negative_cycle, dist);
	free(dist);
	free(utils::mat);

	return 0;
}
